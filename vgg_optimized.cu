/* Includes, system */
#include <cstdio>
#include <iostream>
#include <vector>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>
#include <algorithm>

struct Tensor4d
{
    float *data;
    size_t data_size;

    Tensor4d(int n, int c, int h, int w)
    {
        data_size = n * c * h * w;
        hipMalloc((void **)&data, data_size * sizeof(float));
    }

    ~Tensor4d()
    {
        hipFree(data);
    }
};


struct Bias4d
{
    float *data;
    size_t data_size;

    Bias4d(int n, int c, int h, int w)
    {
        data_size = n * c * h * w;
        float *tmp = (float *)malloc(data_size * sizeof(float));
        for (int i = 0; i < data_size; i++)
        {
            tmp[i] = (float)std::rand() / RAND_MAX / 1000;
        }
        hipMalloc((void **)&data, data_size * sizeof(float));
        hipMemcpy(data, tmp, data_size * sizeof(float), hipMemcpyHostToDevice);
        free(tmp);
    }

    ~Bias4d()
    {
        hipFree(data);
    }
};





struct Filter4d
{
    float *data;
    size_t data_size;

    Filter4d(int n, int c, int h, int w)
    {
        data_size = n * c * h * w;
        float *tmp = (float *)malloc(data_size * sizeof(float));
        for (int i = 0; i < data_size; i++)
        {
            tmp[i] = (float)std::rand() / RAND_MAX / 1000;
        }
        hipMalloc((void **)&data, data_size * sizeof(float));
        hipMemcpy(data, tmp, data_size * sizeof(float), hipMemcpyHostToDevice);
        free(tmp);
    }

    ~Filter4d()
    {
        hipFree(data);
    }
};



struct zeros
{
    float *data;
    size_t data_size;

    zeros(std::vector<int> dims)
    {
        data_size = std::accumulate(dims.begin(), dims.end(), 1, std::multiplies<int>());
        std::vector<float> host_data(data_size, 0.0f);

        hipMalloc((void **)&data, data_size * sizeof(float));
        hipMemcpy(data, host_data.data(), data_size * sizeof(float), hipMemcpyHostToDevice);
    }

    ~zeros()
    {
        hipFree(data);
    }
};



__global__ void reluActivation(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size)
    {
        output[idx] = fmaxf(input[idx], 0.0f);
    }
}


void cuReLU(float *input, float *output, int size)
{
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));
    
    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    
    reluActivation<<<numBlocks, blockSize>>>(d_input, d_output, size);
    
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
}



__global__ void convolution2d(float *input, float *filter, float *output,
                              int input_width, int input_height, int input_channels,
                              int filter_width, int filter_height,
                              int output_width, int output_height,
                              int pad_w, int pad_h,
                              int wstride, int hstride)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z;

    if (idx < output_width && idy < output_height && idz < input_channels)
    {
        int out_idx = idz * output_width * output_height + idy * output_width + idx;

        float value = 0.0f;

        for (int fy = 0; fy < filter_height; ++fy)
        {
            int in_y = idy * hstride + fy - pad_h;
            for (int fx = 0; fx < filter_width; ++fx)
            {
                int in_x = idx * wstride + fx - pad_w;
                if (in_x >= 0 && in_x < input_width && in_y >= 0 && in_y < input_height)
                {
                    int in_idx = idz * input_width * input_height + in_y * input_width + in_x;
                    int filter_idx = idz * filter_width * filter_height + fy * filter_width + fx;
                    value += input[in_idx] * filter[filter_idx];
                }
            }
        }

        output[out_idx] = value;
    }
}



__global__ void addBias(float *output, float *bias, int output_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < output_size)
    {
        output[idx] += bias[idx];
    }
}




void cuConv2D(float *input, float *output, int w, int h, int c, int n, int k,
              int filter_w, int filter_h, int dilation_w, int dilation_h,
              int pad_w, int pad_h, int wstride, int hstride)
{
    // Allocate device memory for input and output
    float *d_input, *d_output;
    hipMalloc((void **)&d_input, n * c * h * w * sizeof(float));
    hipMalloc((void **)&d_output, n * k * h * w * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);

    // Create filter and bias
    Filter4d w_desc(k, c, filter_w, filter_h);
    Bias4d bias(k, 1, 1, 1); // bias is 1x1x1xk

    // Compute output dimensions
    int out_h = (h + 2 * pad_h - dilation_h * (filter_h - 1) - 1) / hstride + 1;
    int out_w = (w + 2 * pad_w - dilation_w * (filter_w - 1) - 1) / wstride + 1;

    // Allocate device memory for output
    Tensor4d h_desc(n, k, out_h, out_w);

    // Compute grid and block dimensions for convolution kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((out_w + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (out_h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   c);

    // Perform convolution
    auto start = std::chrono::steady_clock::now();
    convolution2d<<<numBlocks, threadsPerBlock>>>(d_input, w_desc.data, h_desc.data,
                                                  w, h, c, filter_w, filter_h,
                                                  out_w, out_h, pad_w, pad_h,
                                                  wstride, hstride);

    // Add bias
    addBias<<<(n * k + 255) / 256, 256>>>(h_desc.data, bias.data, n * k);

    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double,
                                    std::micro>(end - start).count());

    std::cout << " " << fwd_time << " ms" << std::endl;


    // Copy output data to host
    hipMemcpy(output, h_desc.data, n * k * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


__global__ void maxPool2d(float *input, float *output,
                          int input_width, int input_height, int input_channels,
                          int pool_width, int pool_height,
                          int output_width, int output_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z;

    if (idx < output_width && idy < output_height && idz < input_channels)
    {
        int out_idx = idz * output_width * output_height + idy * output_width + idx;

        int start_x = idx * pool_width;
        int start_y = idy * pool_height;

        float max_val = -INFINITY;

        for (int fy = 0; fy < pool_height; ++fy)
        {
            int in_y = start_y + fy;
            for (int fx = 0; fx < pool_width; ++fx)
            {
                int in_x = start_x + fx;
                if (in_x < input_width && in_y < input_height)
                {
                    int in_idx = idz * input_width * input_height + in_y * input_width + in_x;
                    max_val = fmaxf(max_val, input[in_idx]);
                }
            }
        }

        output[out_idx] = max_val;
    }
}



void cuMaxPool(float *input, float *output, int w, int h, int c, int n)
{
    // Allocate device memory for input and output
    float *d_input, *d_output;
    hipMalloc((void **)&d_input, n * c * h * w * sizeof(float));
    hipMalloc((void **)&d_output, n * c * (h / 2) * (w / 2) * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);

    // Compute output dimensions
    int out_h = h / 2;
    int out_w = w / 2;

    // Allocate device memory for output
    Tensor4d h_desc(n, c, out_h, out_w);

    // Compute grid and block dimensions for pooling kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((out_w + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (out_h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   c);


    // Perform max pooling

    auto start = std::chrono::steady_clock::now();
    maxPool2d<<<numBlocks, threadsPerBlock>>>(d_input, h_desc.data,
                                              w, h, c, 2, 2, out_w, out_h);

    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double,
                                    std::micro>(end - start).count());

    std::cout << " " << fwd_time << " ms" << std::endl;


    // Copy output data to host
    hipMemcpy(output, h_desc.data, n * c * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


__global__ void matrixVectorMultiply(float *A, float *B, float *C, int left, int right)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < right)
    {
        float sum = 0.0f;
        for (int i = 0; i < left; ++i)
        {
            sum += A[i] * B[i * right + idx];
        }
        C[idx] = sum;
    }
}

void cuFC(float *input, float *output, int left, int right)
{
    // Allocate device memory for input, weights, and output
    float *d_input, *d_weights, *d_output;
    hipMalloc(&d_input, left * sizeof(float));
    hipMalloc(&d_weights, left * right * sizeof(float));
    hipMalloc(&d_output, right * sizeof(float));

    // Copy input and weights from host to device
    hipMemcpy(d_input, input, left * sizeof(float), hipMemcpyHostToDevice);

    float *h_weights = (float *)malloc(left * right * sizeof(float));
    for (int i = 0; i < left * right; i++)
    {
        h_weights[i] = (float)std::rand() / RAND_MAX / 1000;
    }
    hipMemcpy(d_weights, h_weights, left * right * sizeof(float), hipMemcpyHostToDevice);

    // Compute grid and block dimensions
    dim3 threadsPerBlock(256);
    dim3 numBlocks((right + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Perform matrix-vector multiplication
    auto start = std::chrono::steady_clock::now();
    matrixVectorMultiply<<<numBlocks, threadsPerBlock>>>(d_weights, d_input, d_output, left, right);

    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double,
                                    std::micro>(end - start).count());

    std::cout << " " << fwd_time << " ms" << std::endl;


    // Copy output from device to host
    hipMemcpy(output, d_output, right * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);
}



int main()
{
    std::srand(std::time(0));

    float *input;
    float *output;

    int data_size = 224 * 224 * 3 * 1;
    input = (float *)malloc(data_size * sizeof(float));
    for (int i = 0; i < data_size; i++)
    {
        input[i] = (float)std::rand() / RAND_MAX;
    }

    // Block 1
    std::cout << "CONV 224x224x64";
    output = (float *)malloc(224 * 224 * 64 * 1 * sizeof(float));
    cuConv2D(input, output, 224, 224, 3, 1, 64, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 224 * 224 * 64);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 224x224x64";
    output = (float *)malloc(224 * 224 * 64 * 1 * sizeof(float));
    cuConv2D(input, output, 224, 224, 64, 1, 64, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 224 * 224 * 64);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 112x112x64";
    output = (float *)malloc(112 * 112 * 64 * sizeof(float));
    cuMaxPool(input, output, 224, 224, 64, 1);
    std::swap(input, output);
    free(output);

    // Block 2
    std::cout << "CONV 112x112x128";
    output = (float *)malloc(112 * 112 * 128 * 1 * sizeof(float));
    cuConv2D(input, output, 112, 112, 64, 1, 128, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 112 * 112 * 128);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 112x112x128";
    output = (float *)malloc(112 * 112 * 128 * 1 * sizeof(float));
    cuConv2D(input, output, 112, 112, 128, 1, 128, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 112 * 112 * 128);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 56x56x128";
    output = (float *)malloc(56 * 56 * 128 * sizeof(float));
    cuMaxPool(input, output, 112, 112, 128, 1);
    std::swap(input, output);
    free(output);

    // Block 3
    std::cout << "CONV 56x56x256";
    output = (float *)malloc(56 * 56 * 256 * 1 * sizeof(float));
    cuConv2D(input, output, 56, 56, 128, 1, 256, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 56 * 56 * 256);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 56x56x256";
    output = (float *)malloc(56 * 56 * 256 * 1 * sizeof(float));
    cuConv2D(input, output, 56, 56, 256, 1, 256, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 56 * 56 * 256);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 56x56x256";
    output = (float *)malloc(56 * 56 * 256 * 1 * sizeof(float));
    cuConv2D(input, output, 56, 56, 256, 1, 256, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 56 * 56 * 256);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 28x28x256";
    output = (float *)malloc(28 * 28 * 256 * sizeof(float));
    cuMaxPool(input, output, 56, 56, 256, 1);
    std::swap(input, output);
    free(output);

    // Block 4
    std::cout << "CONV 28x28x512";
    output = (float *)malloc(28 * 28 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 28, 28, 256, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 28 * 28 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 28x28x512";
    output = (float *)malloc(28 * 28 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 28, 28, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 28 * 28 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 28x28x512";
    output = (float *)malloc(28 * 28 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 28, 28, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 28 * 28 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * sizeof(float));
    cuMaxPool(input, output, 28, 28, 512, 1);
    std::swap(input, output);
    free(output);

    // Block 5
    std::cout << "CONV 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 14, 14, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 14 * 14 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 14, 14, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 14 * 14 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 14, 14, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 14 * 14 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 7x7x512";
    output = (float *)malloc(7 * 7 * 512 * sizeof(float));
    cuMaxPool(input, output, 14, 14, 512, 1);
    std::swap(input, output);
    free(output);

    // Fully connected layers
    std::cout << "FC 4096";
    output = (float *)malloc(4096 * sizeof(float));
    cuFC(input, output, 7 * 7 * 512, 4096);
    cuReLU(output, output, 4096);
    std::swap(input, output);
    free(output);

    std::cout << "FC 4096";
    output = (float *)malloc(4096 * sizeof(float));
    cuFC(input, output, 4096, 4096);
    cuReLU(output, output, 4096);
    std::swap(input, output);
    free(output);

    std::cout << "FC 1000";
    output = (float *)malloc(1000 * sizeof(float));
    cuFC(input, output, 4096, 1000);

    free(input);
    free(output);

    return 0;
}