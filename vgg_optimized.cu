/* Includes, system */
#include <cstdio>
#include <iostream>
#include <vector>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>
#include <algorithm>

struct Tensor4d
{
    float *data;
    size_t data_size;

    Tensor4d(int n, int c, int h, int w)
    {
        data_size = n * c * h * w;
        hipMalloc((void **)&data, data_size * sizeof(float));
    }

    ~Tensor4d()
    {
        hipFree(data);
    }
};


struct Bias4d
{
    float *data;
    size_t data_size;

    Bias4d(int n, int c, int h, int w)
    {
        data_size = n * c * h * w;
        float *tmp = (float *)malloc(data_size * sizeof(float));
        for (int i = 0; i < data_size; i++)
        {
            tmp[i] = (float)std::rand() / RAND_MAX / 1000;
        }
        hipMalloc((void **)&data, data_size * sizeof(float));
        hipMemcpy(data, tmp, data_size * sizeof(float), hipMemcpyHostToDevice);
        free(tmp);
    }

    ~Bias4d()
    {
        hipFree(data);
    }
};





struct Filter4d
{
    float *data;
    size_t data_size;

    Filter4d(int n, int c, int h, int w)
    {
        data_size = n * c * h * w;
        float *tmp = (float *)malloc(data_size * sizeof(float));
        for (int i = 0; i < data_size; i++)
        {
            tmp[i] = (float)std::rand() / RAND_MAX / 1000;
        }
        hipMalloc((void **)&data, data_size * sizeof(float));
        hipMemcpy(data, tmp, data_size * sizeof(float), hipMemcpyHostToDevice);
        free(tmp);
    }

    ~Filter4d()
    {
        hipFree(data);
    }
};



struct zeros
{
    float *data;
    size_t data_size;

    zeros(std::vector<int> dims)
    {
        data_size = std::accumulate(dims.begin(), dims.end(), 1, std::multiplies<int>());
        std::vector<float> host_data(data_size, 0.0f);

        hipMalloc((void **)&data, data_size * sizeof(float));
        hipMemcpy(data, host_data.data(), data_size * sizeof(float), hipMemcpyHostToDevice);
    }

    ~zeros()
    {
        hipFree(data);
    }
};



__global__ void reluActivation(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size)
    {
        output[idx] = fmaxf(input[idx], 0.0f);
    }
}


void cuReLU(float *input, float *output, int size)
{
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));
    
    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    
    reluActivation<<<numBlocks, blockSize>>>(d_input, d_output, size);
    
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
}



__global__ void convolution2d(float *input, float *filter, float *output,
                              int input_width, int input_height, int input_channels,
                              int filter_width, int filter_height,
                              int output_width, int output_height,
                              int pad_w, int pad_h,
                              int wstride, int hstride)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z;

    if (idx < output_width && idy < output_height && idz < input_channels)
    {
        int out_idx = idz * output_width * output_height + idy * output_width + idx;

        float value = 0.0f;

        for (int fy = 0; fy < filter_height; ++fy)
        {
            int in_y = idy * hstride + fy - pad_h;
            for (int fx = 0; fx < filter_width; ++fx)
            {
                int in_x = idx * wstride + fx - pad_w;
                if (in_x >= 0 && in_x < input_width && in_y >= 0 && in_y < input_height)
                {
                    int in_idx = idz * input_width * input_height + in_y * input_width + in_x;
                    int filter_idx = idz * filter_width * filter_height + fy * filter_width + fx;
                    value += input[in_idx] * filter[filter_idx];
                }
            }
        }

        output[out_idx] = value;
    }
}



__global__ void addBias(float *output, float *bias, int output_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < output_size)
    {
        output[idx] += bias[idx];
    }
}




void cuConv2D(float *input, float *output, int w, int h, int c, int n, int k,
              int filter_w, int filter_h, int dilation_w, int dilation_h,
              int pad_w, int pad_h, int wstride, int hstride)
{
    // Allocate device memory for input and output
    float *d_input, *d_output;
    hipMalloc((void **)&d_input, n * c * h * w * sizeof(float));
    hipMalloc((void **)&d_output, n * k * h * w * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);

    // Create filter and bias
    Filter4d w_desc(k, c, filter_w, filter_h);
    Bias4d bias(k, 1, 1, 1); // bias is 1x1x1xk

    // Compute output dimensions
    int out_h = (h + 2 * pad_h - dilation_h * (filter_h - 1) - 1) / hstride + 1;
    int out_w = (w + 2 * pad_w - dilation_w * (filter_w - 1) - 1) / wstride + 1;

    // Allocate device memory for output
    Tensor4d h_desc(n, k, out_h, out_w);

    // Compute grid and block dimensions for convolution kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((out_w + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (out_h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   c);

    // Perform convolution
    auto start = std::chrono::steady_clock::now();
    convolution2d<<<numBlocks, threadsPerBlock>>>(d_input, w_desc.data, h_desc.data,
                                                  w, h, c, filter_w, filter_h,
                                                  out_w, out_h, pad_w, pad_h,
                                                  wstride, hstride);

    // Add bias
    addBias<<<(n * k + 255) / 256, 256>>>(h_desc.data, bias.data, n * k);

    

    hipDeviceSynchronize();


    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double,
                                    std::micro>(end - start).count());

    std::cout << " " << fwd_time << " ms" << std::endl;
    // Copy output data to host
    hipMemcpy(output, h_desc.data, n * k * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


__global__ void maxPool2d(float *input, float *output,
                          int input_width, int input_height, int input_channels,
                          int pool_width, int pool_height,
                          int output_width, int output_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z;

    if (idx < output_width && idy < output_height && idz < input_channels)
    {
        int out_idx = idz * output_width * output_height + idy * output_width + idx;

        int start_x = idx * pool_width;
        int start_y = idy * pool_height;

        float max_val = -INFINITY;

        for (int fy = 0; fy < pool_height; ++fy)
        {
            int in_y = start_y + fy;
            for (int fx = 0; fx < pool_width; ++fx)
            {
                int in_x = start_x + fx;
                if (in_x < input_width && in_y < input_height)
                {
                    int in_idx = idz * input_width * input_height + in_y * input_width + in_x;
                    max_val = fmaxf(max_val, input[in_idx]);
                }
            }
        }

        output[out_idx] = max_val;
    }
}



void cuMaxPool(float *input, float *output, int w, int h, int c, int n)
{
    // Allocate device memory for input and output
    float *d_input, *d_output;
    hipMalloc((void **)&d_input, n * c * h * w * sizeof(float));
    hipMalloc((void **)&d_output, n * c * (h / 2) * (w / 2) * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);

    // Compute output dimensions
    int out_h = h / 2;
    int out_w = w / 2;

    // Allocate device memory for output
    Tensor4d h_desc(n, c, out_h, out_w);

    // Compute grid and block dimensions for pooling kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((out_w + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (out_h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   c);


    // Perform max pooling

    auto start = std::chrono::steady_clock::now();
    maxPool2d<<<numBlocks, threadsPerBlock>>>(d_input, h_desc.data,
                                              w, h, c, 2, 2, out_w, out_h);

    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double,
                                    std::micro>(end - start).count());

    std::cout << " " << fwd_time << " ms" << std::endl;


    // Copy output data to host
    hipMemcpy(output, h_desc.data, n * c * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


__global__ void matrixVectorMultiply(float *A, float *B, float *C, int left, int right)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < right)
    {
        float sum = 0.0f;
        for (int i = 0; i < left; ++i)
        {
            sum += A[i] * B[i * right + idx];
        }
        C[idx] = sum;
    }
}
__global__ void sgemm_naive(int M, int N, int K, float alpha, const float *A,
                            const float *B, float beta, float *C) {
  // compute position in C that this thread is responsible for
  const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < M && y < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[x * K + i] * B[i * N + y];
    }
    // C = α*(A@B)+β*C
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }
}

// void cuFC(float *input, float *output, int left, int right)
// {
//     // Allocate device memory for input, weights, and output
//     float *d_input, *d_weights, *d_output;
//     cudaMalloc(&d_input, left * sizeof(float));
//     cudaMalloc(&d_weights, left * right * sizeof(float));
//     cudaMalloc(&d_output, right * sizeof(float));

//     // Copy input and weights from host to device
//     cudaMemcpy(d_input, input, left * sizeof(float), cudaMemcpyHostToDevice);

//     float *h_weights = (float *)malloc(left * right * sizeof(float));
//     for (int i = 0; i < left * right; i++)
//     {
//         h_weights[i] = (float)std::rand() / RAND_MAX / 1000;
//     }
//     cudaMemcpy(d_weights, h_weights, left * right * sizeof(float), cudaMemcpyHostToDevice);

//     // Compute grid and block dimensions
//     dim3 threadsPerBlock(256);
//     dim3 numBlocks((right + threadsPerBlock.x - 1) / threadsPerBlock.x);

//     // Perform matrix-vector multiplication
//     auto start = std::chrono::steady_clock::now();
//     matrixVectorMultiply<<<numBlocks, threadsPerBlock>>>(d_weights, d_input, d_output, left, right);

//     auto end = std::chrono::steady_clock::now();
//     int fwd_time = static_cast<int>(std::chrono::duration<double,
//                                     std::micro>(end - start).count());

//     std::cout << " " << fwd_time << " ms" << std::endl;


//     // Copy output from device to host
//     cudaMemcpy(output, d_output, right * sizeof(float), cudaMemcpyDeviceToHost);

//     // Free device memory
//     cudaFree(d_input);
//     cudaFree(d_weights);
//     cudaFree(d_output);
// }
void cuFC(float *input, float *output, int left, int right)
{
    int lda = 1, ldb = left, ldc = 1, m = 1, k = left, n = right;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    float *h_B = (float *)malloc(left * right * sizeof(float));
    for (int i = 0; i < left * right; i++)
    {
        h_B[i] = (float)std::rand() / RAND_MAX / 1000;
    }

    // Allocate 3 arrays on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, left * sizeof(float));
    hipMalloc(&d_B, left * right * sizeof(float));
    hipMalloc(&d_C, right * sizeof(float));

    hipMemcpy(d_A, input, left * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, left * right * sizeof(float), hipMemcpyHostToDevice);



    auto start = std::chrono::steady_clock::now();

    // Do the actual multiplication
    sgemm_naive<<<64, 64>>>(m, n, k, 1.0, d_A, d_B, 0.0, d_C);
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double,
                                    std::micro>(end - start).count());

    std::cout << " " << fwd_time << " ms" << std::endl;

    hipMemcpy(output, d_C, right * sizeof(float), hipMemcpyDeviceToHost);
    // Destroy the handle
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(cudaError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != cudaSuccess) 
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", cudaGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }



// void cuFC(float *input, float *output, int left, int right)
// {
//     float *d_input, *d_weights, *d_output;
//     gpuErrchk(cudaMalloc(&d_input, left * sizeof(float)));
//     gpuErrchk(cudaMalloc(&d_weights, left * right * sizeof(float)));
//     gpuErrchk(cudaMalloc(&d_output, right * sizeof(float)));

//     gpuErrchk(cudaMemcpy(d_input, input, left * sizeof(float), cudaMemcpyHostToDevice));

//     float *h_weights = (float *)malloc(left * right * sizeof(float));
//     for (int i = 0; i < left * right; i++) {
//         h_weights[i] = (float)std::rand() / RAND_MAX / 1000;
//     }
//     gpuErrchk(cudaMemcpy(d_weights, h_weights, left * right * sizeof(float), cudaMemcpyHostToDevice));
//     free(h_weights);

//     dim3 threadsPerBlock(256);
//     dim3 numBlocks((right + threadsPerBlock.x - 1) / threadsPerBlock.x);

//     auto start = std::chrono::steady_clock::now();
//     matrixVectorMultiply<<<numBlocks, threadsPerBlock>>>(d_weights, d_input, d_output, left, right);
//     gpuErrchk(cudaPeekAtLastError());
//     gpuErrchk(cudaDeviceSynchronize()); // Ensure the kernel completes and captures any errors

//     auto end = std::chrono::steady_clock::now();
//     std::chrono::duration<double, std::micro> elapsed = end - start;
//     std::cout << "Kernel Execution Time: " << elapsed.count() << " us" << std::endl;

//     gpuErrchk(cudaMemcpy(output, d_output, right * sizeof(float), cudaMemcpyDeviceToHost));

//     cudaFree(d_input);
//     cudaFree(d_weights);
//     cudaFree(d_output);
// }



int main()
{
    std::srand(std::time(0));

    float *input;
    float *output;

    int data_size = 224 * 224 * 3 * 1;
    input = (float *)malloc(data_size * sizeof(float));
    for (int i = 0; i < data_size; i++)
    {
        input[i] = (float)std::rand() / RAND_MAX;
    }


    
    // Block 1
    std::cout << "CONV 224x224x64";
    output = (float *)malloc(224 * 224 * 64 * 1 * sizeof(float));
    cuConv2D(input, output, 224, 224, 3, 1, 64, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 224 * 224 * 64);

    std::swap(input, output);
    free(output);

    std::cout << "CONV 224x224x64";
    output = (float *)malloc(224 * 224 * 64 * 1 * sizeof(float));
    cuConv2D(input, output, 224, 224, 64, 1, 64, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 224 * 224 * 64);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 112x112x64";
    output = (float *)malloc(112 * 112 * 64 * sizeof(float));
    cuMaxPool(input, output, 224, 224, 64, 1);
    std::swap(input, output);
    free(output);

    // Block 2
    std::cout << "CONV 112x112x128";
    output = (float *)malloc(112 * 112 * 128 * 1 * sizeof(float));
    cuConv2D(input, output, 112, 112, 64, 1, 128, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 112 * 112 * 128);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 112x112x128";
    output = (float *)malloc(112 * 112 * 128 * 1 * sizeof(float));
    cuConv2D(input, output, 112, 112, 128, 1, 128, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 112 * 112 * 128);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 56x56x128";
    output = (float *)malloc(56 * 56 * 128 * sizeof(float));
    cuMaxPool(input, output, 112, 112, 128, 1);
    std::swap(input, output);
    free(output);

    // Block 3
    std::cout << "CONV 56x56x256";
    output = (float *)malloc(56 * 56 * 256 * 1 * sizeof(float));
    cuConv2D(input, output, 56, 56, 128, 1, 256, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 56 * 56 * 256);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 56x56x256";
    output = (float *)malloc(56 * 56 * 256 * 1 * sizeof(float));
    cuConv2D(input, output, 56, 56, 256, 1, 256, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 56 * 56 * 256);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 56x56x256";
    output = (float *)malloc(56 * 56 * 256 * 1 * sizeof(float));
    cuConv2D(input, output, 56, 56, 256, 1, 256, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 56 * 56 * 256);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 28x28x256";
    output = (float *)malloc(28 * 28 * 256 * sizeof(float));
    cuMaxPool(input, output, 56, 56, 256, 1);
    std::swap(input, output);
    free(output);

    // Block 4
    std::cout << "CONV 28x28x512";
    output = (float *)malloc(28 * 28 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 28, 28, 256, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 28 * 28 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 28x28x512";
    output = (float *)malloc(28 * 28 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 28, 28, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 28 * 28 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 28x28x512";
    output = (float *)malloc(28 * 28 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 28, 28, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 28 * 28 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * sizeof(float));
    cuMaxPool(input, output, 28, 28, 512, 1);
    std::swap(input, output);
    free(output);

    // Block 5
    std::cout << "CONV 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 14, 14, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 14 * 14 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 14, 14, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 14 * 14 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "CONV 14x14x512";
    output = (float *)malloc(14 * 14 * 512 * 1 * sizeof(float));
    cuConv2D(input, output, 14, 14, 512, 1, 512, 3, 3, 1, 1, 1, 1, 1, 1);
    cuReLU(output, output, 14 * 14 * 512);
    std::swap(input, output);
    free(output);

    std::cout << "POOLMAX 7x7x512";
    output = (float *)malloc(7 * 7 * 512 * sizeof(float));
    cuMaxPool(input, output, 14, 14, 512, 1);
    std::swap(input, output);
    free(output);

    // Fully connected layers
    std::cout << "FC 4096";
    output = (float *)malloc(4096 * sizeof(float));
    cuFC(input, output, 7 * 7 * 512, 4096);
    cuReLU(output, output, 4096);
    std::swap(input, output);
    free(output);

    std::cout << "FC 4096";
    output = (float *)malloc(4096 * sizeof(float));
    cuFC(input, output, 4096, 4096);
    cuReLU(output, output, 4096);
    std::swap(input, output);
    free(output);

    std::cout << "FC 1000";
    output = (float *)malloc(1000 * sizeof(float));
    cuFC(input, output, 4096, 1000);

    free(input);
    free(output);

    return 0;
}